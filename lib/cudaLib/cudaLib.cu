#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h> 
#include <iostream>

#include "hip/hip_runtime.h"
#include ""
#include "cudaLib.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


float timing_experiment(kernelCall func) {
    float time_ms = 0.0;
    int N_rep = 5;
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(start, 0);
    for (int i = 0; i < N_rep; ++i) {
        func();
    }
    hipEventRecord(end, 0);
    hipDeviceSynchronize();
    hipEventElapsedTime(&time_ms, start, end);
    time_ms /= N_rep;
    
    hipEventDestroy(start);
    hipEventDestroy(end);
    return time_ms;
}

__global__ void checkId(){
	printf("threadIdx: (%d, %d, %d)	blockIdx: (%d, %d, %d)	blockDim: (%d, %d, %d) gridDim: (%d, %d, %d)\n", 
	threadIdx.x, threadIdx.y, threadIdx.z, blockIdx.x, blockIdx.y, blockIdx.z,
	blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z
	);
}

void printDeviceInfo() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (KHz): %d\n",
            prop.memoryClockRate);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
            2.0 * prop.memoryClockRate * (prop.memoryBusWidth / 8) / 1.0e6);
    }
}

__global__ void general2DConvolution(float* d_in, float* d_out, int width, int height, float* d_kernel, int kernelWidth, int kernelHeight) {
    int rx = kernelWidth / 2, ry = kernelHeight / 2;
    for (int iy = blockIdx.y * blockDim.y + threadIdx.y; iy < height; iy += blockDim.y * gridDim.y) {
        for (int ix = blockIdx.x * blockDim.x + threadIdx.x; ix < width; ix += blockDim.x * gridDim.x) {
            float value = 0.0;
            for (int yshift = -ry; yshift <= ry; ++yshift) {
                for (int xshift = -rx; xshift <= rx; ++xshift) {
                    if (ix + xshift >= 0 && ix + xshift < width && iy + yshift >= 0 && iy + yshift < height) {
                        value += d_in[(iy + yshift) * width + (ix + xshift)] * d_kernel[(ry + yshift) * kernelWidth + (rx + xshift)];
                    }
                }
            }
            d_out[iy * width + ix] = value;
        }
    }
}

void Convolution2D(float* h_in, float* h_out, int width, int height, float* h_kernel, int kernelWidth, int kernelHeight) {
    if (!(kernelWidth % 2 == 1 && kernelHeight % 2 == 1)) {
        std::cout << "Invaid input: Input kernel must have odd size!" << std::endl;
        return;
    }
    float* d_in, * d_out, * d_kernel;
    int imageSizeInByte = width * height * sizeof(float), kernelSizeInByte = kernelWidth * kernelHeight * sizeof(float);
    //allocate device memory
    gpuErrchk(hipMalloc((void**)&d_in, imageSizeInByte));
    gpuErrchk(hipMalloc((void**)&d_out, imageSizeInByte));
    gpuErrchk(hipMalloc((void**)&d_kernel, kernelSizeInByte));
    //copy data from host to device
    gpuErrchk(hipMemcpy(d_in, h_in, imageSizeInByte, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_out, h_out, imageSizeInByte, hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_kernel, h_kernel, kernelSizeInByte, hipMemcpyHostToDevice));
    //call the kernel
    dim3 blockSize(32, 32);
    dim3 gridSize(16, 16);
    general2DConvolution << <gridSize, blockSize >> > (d_in, d_out, width, height, d_kernel, kernelWidth, kernelHeight);
    //copy data back
    gpuErrchk(hipMemcpy(h_out, d_out, imageSizeInByte, hipMemcpyDeviceToHost));
    //free memory
    gpuErrchk(hipFree(d_in));
    gpuErrchk(hipFree(d_out));
    gpuErrchk(hipFree(d_kernel));
}

